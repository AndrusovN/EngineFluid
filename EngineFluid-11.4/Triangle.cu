#include "Triangle.cuh"
#include "assert.cuh"
#include <new>

__host__ __device__ bool Triangle::isInsideTriangle(Vector3 point) const
{
	return ((_vertices[1] - _vertices[0]).angle_sin(point - _vertices[0]) >= 0) &&
		((_vertices[2] - _vertices[1]).angle_sin(point - _vertices[1]) >= 0) &&
		((_vertices[0] - _vertices[2]).angle_sin(point - _vertices[2]) >= 0);
}

__host__ __device__ Triangle::Triangle(Vector3 a, Vector3 b, Vector3 c) :
	_vertices{ a, b, c },
	_center((a + b + c) / 3),
	_normal((b - a).cross(b - c)) {}

__host__ __device__ Triangle::Triangle() :
	_vertices{Vector3::ZERO(), Vector3::ZERO(), Vector3::ZERO()}, 
	_center(Vector3::ZERO()), 
	_normal(Vector3::ZERO())  {}

__host__ __device__ const Triangle Triangle::operator=(const Triangle& other)
{
	new (this) Triangle(other);
	return *this;
}

__host__ __device__ bool Triangle::operator==(const Triangle& other) const
{
	for (int offset = 0; offset < 3; offset++)
	{
		bool ok = _vertices[0] == other.get_vertex(offset) &&
			_vertices[1] == other.get_vertex((offset + 1) % 3) &&
			_vertices[2] == other.get_vertex((offset + 2) % 3);
		if (ok) {
			return true;
		}
	}
	return false;
}

__host__ __device__ bool Triangle::operator!=(const Triangle& other) const
{
	return !(*this == other);
}

__host__ __device__ const Vector3 Triangle::rayIntersection(const Vector3& startPoint, const Vector3& direction) const
{
	// formula from https://en.wikipedia.org/wiki/Line%E2%80%93plane_intersection
	number_t d = (_vertices[0] - startPoint).dot(_normal) / (direction.dot(_normal));
	Vector3 point = direction * d + startPoint;
	if (isInsideTriangle(point) && d > 0) {
		return point;
	}
	else {
		return Vector3::INFINITY_VECTOR();
	}
}

__host__ __device__ bool Triangle::isInside(const Vector3& other) const
{
	return _normal.dot(other - _center) <= 0;
}

__host__ __device__ const Vector3 Triangle::normal() const
{
	return _normal;
}

__host__ __device__ const Triangle Triangle::reversed() const
{
	return Triangle(_vertices[0], _vertices[2], _vertices[1]);
}

__host__ __device__ const Vector3 Triangle::get_vertex(int index) const
{
	assert(0 <= index && index <= 3);
	return _vertices[index];
}
