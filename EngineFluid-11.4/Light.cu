#include "hip/hip_runtime.h"
#define NOMINMAX
#include "Light.cuh"

#define min(a, b) ((a) < (b) ? (a) : (b))
#define max(a, b) ((a) < (b) ? (b) : (a))

__host__ __device__ EngineColor::EngineColor()
{
	r = 0;
	g = 0;
	b = 0;
	a = 1;
}

__host__ __device__ EngineColor::EngineColor(_byte r, _byte g, _byte b, _byte a)
{
	this->r = r;
	this->g = g;
	this->b = b;
	this->a = a;
}

__host__ __device__ EngineColor::EngineColor(Color base)
{
	// windows.h COLORREF (a.k.a. Color there) is actually unsigned long long
	// so it stores data like this:
	// | 40 bites empty | 8 bites blue color | 8 bites green color | 8 bites red color |
	unsigned long colorUnwrapped = base;
	r = colorUnwrapped % 256;
	g = (colorUnwrapped >> 8) % 256;
	b = (colorUnwrapped >> 16) % 256;
	a = 1;
}

__host__ __device__ EngineColor EngineColor::operator=(const EngineColor& other)
{
	r = other.r;
	g = other.g;
	b = other.b;
	a = other.a;

	return *this;
}

__host__ __device__ const EngineColor EngineColor::operator+(const EngineColor& other) const
{
	int _r = (int)r + other.r;
	int _b = (int)b + other.b;
	int _g = (int)g + other.g;
	int _a = (int)a + other.a;
	_a = max(1, _a);

	return EngineColor((_r * 255) / _a, (_g * 255) / _a, (_b * 255) / _a, min(a, 255));
}

__host__ __device__ const EngineColor EngineColor::operator==(const EngineColor& other) const
{
	return r == other.r &&
		g == other.g &&
		b == other.b && 
		a == other.a;
}

__host__ __device__ Color EngineColor::toWinColor()
{
	return RGB(r, g, b);
}


__host__ __device__ int GeneralLight::typeId() const
{
	return GENERAL_LIGHT_TYPEID;
}

__host__ __device__ GeneralLight::GeneralLight(GameObject* parent, EngineColor lightColor) : Component(parent)
{
	_lightColor = lightColor;
}

__host__ __device__ void GeneralLight::awake()
{
	_transform = gameObject()->getComponentOfType<Transform>();
}

__host__ __device__ EngineColor GeneralLight::getLight(Vector3 normal)
{
	int angle_parameter = max(normal.angle_cos(_transform->forward()), 0) * 255;

	EngineColor e = _lightColor;
	e.a = angle_parameter;
	e = e + EngineColor(0, 0, 0, 1);
	return e;
}

__host__ void GeneralLight::moveToDevice()
{
}

__host__ void GeneralLight::moveToHost()
{
}

__host__ __device__ void GeneralLight::resetGameObject(GameObject* object)
{
	Component::resetGameObject(object);
	_transform = object->getComponentOfType<Transform>();
}
