#include "GameManager.cuh"
#include <assert.h>

GameManager* GameManager::_instance = nullptr;

GameManager::GameManager(UIDrawer* drawer, std::vector<Scene*> scenes)
{
	assert(_instance == nullptr);

	_instance = this;
	this->drawer = drawer;
	_scenes = scenes;
	_currentSceneId = 0;
}

void GameManager::addVisualEffect(VisualEffect* effect)
{
	assert(effect != nullptr);

	_visualEffects.push_back(effect);
}

void GameManager::removeVisualEffect(VisualEffect* effect)
{
	for (int i = 0; i < _visualEffects.size(); i++)
	{
		if ((void*)_visualEffects[i] == (void*)effect) {
			_visualEffects.erase(_visualEffects.begin() + i);
			return;
		}
	}
}

GameManager* GameManager::instance()
{
	return _instance;
}

Scene* GameManager::getScene(int index)
{
	assert(0 <= index && index < _scenes.size());

	return _scenes[index];
}

void GameManager::stop()
{
	_stopped = true;
}

void GameManager::run()
{
	assert(scenes.size() > 0);

	changeScene(0);

	while (!_stopped) {
		_fieldsMutex.lock();
		for (auto component : _scenes[_currentSceneId]->components())
		{
			component->update();
		}
		for (auto component : _scenes[_currentSceneId]->components())
		{
			component->lateUpdate();
		}
		_fieldsMutex.unlock();

		Color* colorMap = drawer->getColorMap();
		int width = drawer->getWidth();
		int height = drawer->getHeight();

		for (auto effect : _visualEffects)
		{
			effect->apply(colorMap, width, height, _scenes[_currentSceneId]);
		}

		drawer->display();
	}
}

void GameManager::changeScene(int sceneIndex)
{
	assert(0 <= sceneIndex && sceneIndex < _scenes.size());

	_fieldsMutex.lock();

	_currentSceneId = sceneIndex;

	for (auto component : _scenes[_currentSceneId]->components())
	{
		component->awake();
	}

	for (auto component : _scenes[_currentSceneId]->components())
	{
		component->start();
	}

	_fieldsMutex.unlock();
}
