#include "hip/hip_runtime.h"
#include "Camera.cuh"
#include "GameManager.cuh"
#include "Mesh.cuh"
#include "Light.cuh"

#include "hip/hip_runtime.h"
#include ""
#include "assert.cuh"

__device__ Pair<Vector3, Triangle> rayCastGetTriangle(Vector3 startPoint, Vector3 direction, Scene* scene)
{
	Scene* current = scene;

	Triangle empty = Triangle(Vector3::ZERO(), Vector3::ZERO(), Vector3::ZERO());

	Vector3 nearestPoint = Vector3::INFINITY_VECTOR();
	Triangle nearestTriangle = Triangle(Vector3::ZERO(), Vector3::ZERO(), Vector3::ZERO());

	Vector<Mesh>* meshes = current->getComponents<Mesh>();

	for (int i = 0; i < meshes->size(); i++)
	{
		Mesh* mesh = (*meshes)[i];
		for (int triangleIndex = 0; triangleIndex < mesh->size(); triangleIndex++)
		{
			Triangle t = mesh->get_triangle(triangleIndex);

			Vector3 intersection = t.rayIntersection(startPoint, direction);

			if (Vector3::sqrDistance(intersection, startPoint) < Vector3::sqrDistance(nearestPoint, startPoint)) {
				nearestPoint = intersection;
				nearestTriangle = t;
			}
		}
	}

	delete meshes;

	return { nearestPoint, nearestTriangle };
}

__device__ EngineColor rayCast(Vector3 startPoint, Vector3 direction, Scene* scene)
{
	Scene* current = scene;

	Triangle empty = Triangle(Vector3::ZERO(), Vector3::ZERO(), Vector3::ZERO());

	auto pointNtriangle = rayCastGetTriangle(startPoint, direction, scene);

	Vector3 nearestPoint = pointNtriangle.first;
	Triangle nearestTriangle = pointNtriangle.second;

	if (nearestTriangle == empty) {
		return EngineColor(0, 0, 0);
	}

	EngineColor result = EngineColor(0, 0, 0, 1);

	Vector<GeneralLight>* lights = current->getComponents<GeneralLight>();

	for (int i = 0; i < lights->size(); i++)
	{
		GeneralLight* light = (*lights)[i];
		Vector3 lightPosition = light->gameObject()->getComponentOfType<Transform>()->position();

		if (nearestTriangle.normal().angle_cos(lightPosition - nearestPoint) > 0) {
			auto pointNTriangleLight = rayCastGetTriangle(nearestPoint, lightPosition - nearestPoint, scene);
			if (Vector3::sqrDistance(nearestPoint, lightPosition) < Vector3::sqrDistance(nearestPoint, pointNTriangleLight.first)) {
				result = result + light->getLight(nearestTriangle.normal());
			}
		}
	}

	delete lights;

	return result;
}

__global__ void castRays(EngineColor* map, Vector3 xPixelVector, Vector3 yPixelVector, 
	Vector3 position, Vector3 forward, number_t minRenderDistance, number_t maxRenderDistance, Scene* scene)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	int width = blockDim.y;
	int height = blockDim.x;

	// center the zero point
	x -= width / 2;
	y -= height / 2;
	y = -y;

	
	Vector3 direction = forward + xPixelVector * x + yPixelVector * y;
	Vector3 startPoint = position + direction * minRenderDistance;

	map[x * height + y] = rayCast(startPoint, direction, scene);
}

__host__ void Camera::renderColorsOnCUDA()
{
	int width = renderSpace.x2 - renderSpace.x1;
	int height = renderSpace.y2 - renderSpace.y1;

	EngineColor* device_map;
	hipMalloc((void**)&device_map, height * width * sizeof(EngineColor));

	Vector3 xPixel = _transform->right() * tanf(_angleX / 2) * 2;
	Vector3 yPixel = _transform->up() * tanf(_angleY / 2) * 2;


	castRays<<< width, height >>>(device_map, xPixel, yPixel, _transform->position(), _transform->forward(), 
		_minRenderDistance, _maxRenderDistance, GameManager::instance()->getCurrentScene());

	EngineColor* map = new EngineColor[width * height];
	hipMemcpy(map, device_map, height * width* sizeof(EngineColor), hipMemcpyDeviceToHost);

	Color* total_map = new Color[ _drawer->getHeight() * _drawer->getWidth() ];
	Color* oldMap = _drawer->getColorMap();
	for (int i = 0; i < _drawer->getWidth(); i++)
	{
		for (int j = 0; j < _drawer->getHeight(); j++)
		{
			int x = i - renderSpace.x1;
			int y = j - renderSpace.y1;
			if (0 <= x && x < width && 0 <= y && y < height) {
				total_map[i * _drawer->getHeight() + j] = map[x * height + y].toWinColor();
			}
			else {
				total_map[i * _drawer->getHeight() + j] = oldMap[i * _drawer->getHeight() + j];
			}
		}
	}

	_drawer->resetColorMap(total_map);
}

__host__ __device__ int Camera::typeId() const 
{
	return CAMERA_TYPE_ID;
}

__host__ __device__ Camera::Camera(GameObject* parent) : Component(parent)
{
	_drawer = nullptr;
	_angleX = 0;
	_angleY = 0;
	_minRenderDistance = 0;
	_maxRenderDistance = 1;
	_transform = nullptr;
}

__host__ __device__ Camera::Camera(GameObject* parent, UIDrawer* drawer, rect renderRect, number_t angleX, number_t angleY) : Component(parent), _drawer(drawer)
{
	assert(angleX > 0);
	assert(angleY > 0);

	renderSpace = renderRect;
	_angleX = angleX;
	_angleY = angleY;
}

__host__ __device__ Camera::Camera(GameObject* parent, 
	UIDrawer* drawer, rect renderRect, number_t angleX, number_t angleY, 
	number_t minRenderDistance, number_t maxRenderDistance) : Component(parent), _drawer(drawer)
{
	assert(angleX > 0);
	assert(angleY > 0);
	assert(minRenderDistance > 0);
	assert(maxRenderDistance > 0);

	renderSpace = renderRect;
	_angleY = angleY / 180 * PI();
	_angleX = angleX / 180 * PI();
	_minRenderDistance = minRenderDistance;
	_maxRenderDistance = maxRenderDistance;
}

__host__ __device__ void Camera::awake()
{
	_transform = gameObject()->getComponentOfType<Transform>();
	assert(_transform != nullptr);
}

__host__ void Camera::deviceUpdate()
{
	renderColorsOnCUDA();
}

__host__ void Camera::moveToDevice()
{
}

__host__ void Camera::moveToHost()
{
}

__host__ __device__ void Camera::resetGameObject(GameObject* object)
{
	Component::resetGameObject(object);
	_transform = object->getComponentOfType<Transform>();
}
