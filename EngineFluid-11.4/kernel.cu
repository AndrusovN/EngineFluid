#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "Drawing.h"

#include <stdio.h>
#include <Windows.h>
#include <thread>

const int WIDTH = 1024;
const int HEIGHT = 700;

hipError_t renderWithCuda(Color *map, int time);
int main(UIDrawer* drawer);

__global__ void render(Color* map, int time)
{
    int x = threadIdx.x;
    int y = blockIdx.x;

    int red = 140 + (int)(110.0f * sinf((float)(x + time / 10) / 50.0f));
    int green = 140 + (int)(110.0f * sinf((float)(y + time / 5) / 100.0f));
    map[x + y * blockDim.x] = RGB(red, green, 30);
}

int WINAPI wWinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, PWSTR pCmdLine, int nCmdShow)
{
    UIDrawer* drawer = new UIDrawer(WIDTH, HEIGHT, "Nice drawer!", hInstance);

    std::thread mainThread(&main, drawer);
    mainThread.detach();

    drawer->processWindowEventsLoop();

    return 0;
}

int main(UIDrawer* drawer)
{
    int time = 0;
    while (true) {
        Color* map = new Color[WIDTH * HEIGHT];

        hipError_t cudaStatus = renderWithCuda(map, time);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addWithCuda failed!");
            return 1;
        }

        drawer->resetColorMap(map);
        drawer->display();

        time++;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t renderWithCuda(Color* map, int time)
{
    Color* device_map = NULL;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&device_map, HEIGHT * WIDTH * sizeof(Color));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    render<<<HEIGHT, WIDTH>>>(device_map, time);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(map, device_map, HEIGHT * WIDTH * sizeof(Color), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(device_map);
    
    return cudaStatus;
}
